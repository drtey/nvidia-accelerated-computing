#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

__global__ void bodyForce(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; 
        float Fy = 0.0f; 
        float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;
            Fx += dx * invDist3; 
            Fy += dy * invDist3; 
            Fz += dz * invDist3;
        }

        p[i].vx += dt*Fx; 
        p[i].vy += dt*Fy; 
        p[i].vz += dt*Fz;
    }
}

__global__ void integratePositions(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        p[i].x += p[i].vx*dt;
        p[i].y += p[i].vy*dt;
        p[i].z += p[i].vz*dt;
    }
}

int main(const int argc, const char** argv) {
    int nBodies = 2<<11;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    const char * initialized_values;
    const char * solution_values;
    
    if (nBodies == 2<<11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }
    
    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f;
    const int nIters = 10;
    int bytes = nBodies * sizeof(Body);
    
    // Allocate unified memory
    Body *p;
    hipMallocManaged(&p, bytes);

    read_values_from_file(initialized_values, (float*)p, bytes);

    // Use 256 threads per block for good occupancy
    int threadsPerBlock = 256;
    int blocksPerGrid = (nBodies + threadsPerBlock - 1) / threadsPerBlock;

    double totalTime = 0.0;

    // Prefetch data to GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(p, bytes, device, NULL);

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        bodyForce<<<blocksPerGrid, threadsPerBlock>>>(p, dt, nBodies);
        hipDeviceSynchronize();

        integratePositions<<<blocksPerGrid, threadsPerBlock>>>(p, dt, nBodies);
        hipDeviceSynchronize();

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

    // Prefetch back to CPU for file writing
    hipMemPrefetchAsync(p, bytes, hipCpuDeviceId, NULL);
    write_values_to_file(solution_values, (float*)p, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    hipFree(p);
    return 0;
}